#include "hip/hip_runtime.h"
#include "lodepng.h"

#include <iostream>
#include <vector>
#include <algorithm>
#include <cassert>

template<typename T>
class Grid {
public:
    enum State {Host, GPU} state;
    unsigned int width, height;

    __host__ Grid(unsigned int w, unsigned int h) : width(w), height(h), state(Host) {
        hipMalloc(&vals_device, size());
        vals_host = (T*) calloc(length(), sizeof(T));
    }

    __host__ __device__ Grid(T* vh, T* vd, unsigned int w, unsigned int h, State st = Host) : vals_host(vh), vals_device(vd), width(w), height(h), state(st) {}

    __host__ __device__ ~Grid() {
#ifndef __CUDA_ARCH__
        free(vals_host);
        hipFree(vals_device);
#endif
    }

    __host__ __device__ T& operator[](int index) {
#ifdef __CUDA_ARCH__
        assert (state == GPU);
        return vals_device[index];
#else
        assert (state == Host);
        return vals_host[index];
#endif
    }

    __host__ __device__ const T& operator[](int index) const {
#ifdef __CUDA_ARCH__
        assert (state == GPU);
        return vals_device[index];
#else
        assert (state == Host);
        return vals_host[index];
#endif
    } 

    __host__ __device__ T& at(int index_row, int index_column) {
        return this[index_row * width + index_column];
    }

    __host__ __device__ T& at(int index_row, int index_column) const {
        return this[index_row * width + index_column];
    }

    __host__ __device__ unsigned int length() const {
        return width * height;
    }

    __host__ __device__ unsigned int size() const {
        return width * height * sizeof(T);
    }

    __host__ T* begin() const {
        assert (state == Host);
        return vals_host;
    }

    __host__ T* end() const {
        assert (state == Host);
        return vals_host + (width * height);
    }

private:
    T *vals_host;
    T *vals_device;
};

unsigned int rescale(unsigned int val, unsigned int data_max, unsigned int new_max) {
    float scaled = (float)val * (float)new_max / (float)data_max;
    return (unsigned int)scaled;
}

void serialize(const char *filename, const Grid<unsigned int>& data)
{
    std::vector<unsigned char> raw_data;
    raw_data.reserve(data.width * data.height * 4);

    unsigned int max = *std::max_element(data.begin(), data.end());
    for (int i = 0; i < data.width * data.height; i++) {
        unsigned char byte = rescale(data[i], max, 255);
        for (int channel = 0; channel < 3; channel++)
            raw_data.push_back(byte);
        raw_data.push_back(255);
    }

    unsigned error = lodepng::encode(filename, raw_data, data.width, data.height);
    if (error)
        std::cout << "encoder error " << error << ": " << lodepng_error_text(error) << std::endl;
}

int main()
{
    Grid<unsigned int> h(10, 10);

    serialize("tmp.png", h);
    return 0;
}
